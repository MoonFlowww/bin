#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "kernel.cuh"

#include <cstdio>

// --------------------------------- LINEAR ---------------------------------------

__global__ void addition(float* X, float* B, float* R, int rows, int cols) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < cols && row < rows) {
        R[row * cols + col] = X[row * cols + col] + B[row * cols + col];
    }
}

__global__ void subtraction(float* A, float* B, float* C, int rows, int cols) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < cols && row < rows) {
        C[row * cols + col] = A[row * cols + col] - B[row * cols + col];
    }
}




// --------------------------------- NON LINEAR ---------------------------------------

__global__ void multiplication(float* X, float* W, float* R, int m, int k, int n) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < n && row < m) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            sum += X[row * k + i] * W[i * n + col];
        }
        R[row * n + col] = sum;
    }
}

__global__ void division(float* X, float* W, float* R, int m, int k, int n) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < n && row < m) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            sum += X[row * k + i] / W[i * n + col];
        }
        R[row * n + col] = sum;
    }
}


