#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#include "CUDA_HUB.cuh"
#include "kernel.cuh"

inline void HANDLE_ERROR(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}

int getCoresPerSM(int major, int minor) {
    // Defines the number of CUDA cores per multiprocessor for different architectures
    struct SMtoCores {
        int SM;
        int Cores;
    };

    SMtoCores nGpuArchCoresPerSM[] = {
        {0x30, 192}, // Kepler
        {0x32, 192}, // Kepler
        {0x35, 192}, // Kepler
        {0x37, 192}, // Kepler
        {0x50, 128}, // Maxwell
        {0x52, 128}, // Maxwell
        {0x53, 128}, // Maxwell
        {0x60, 64},  // Pascal
        {0x61, 128}, // Pascal
        {0x62, 128}, // Pascal
        {0x70, 64},  // Volta
        {0x72, 64},  // Volta
        {0x75, 64},  // Turing
        {0x80, 64},  // Ampere
        {0x86, 128}, // Ampere
        {0x90, 128}, // Ada Lovelace (RTX 40 series)
        {-1, -1}
    };

    int index = 0;
    while (nGpuArchCoresPerSM[index].SM != -1) {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
            return nGpuArchCoresPerSM[index].Cores;
        }
        index++;
    }
    //std::cerr << "Unknown GPU architecture" << std::endl;
    return -1;
}

int Devices() {
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    HANDLE_ERROR(error_id);
    if (deviceCount == 0)
        printf("There are no available device(s) that support CUDA\n");
    else {
        
        std::cout << "\n\033[1m\033[37m*~~~~~~~~~~~~~~GPUs~~~~~~~~~~~~~*\033[0m" << std::endl;
        for (int GPU = 0; GPU < deviceCount; ++GPU) {
            hipSetDevice(GPU);
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, GPU);
            int coresPerSM = getCoresPerSM(deviceProp.major, deviceProp.minor);
            int totalCores = coresPerSM > 0 ? coresPerSM * deviceProp.multiProcessorCount : 0;
            std::cout << "  \033[1m\033[37m" << GPU + 1 << "> " << deviceProp.name << "\033[0m\n"
                << "     |-> \033[1m\033[37m" << deviceProp.multiProcessorCount << "\033[0m Multi-Processors\n"
                << "     |-> ";
            if (totalCores > 0) {
                std::cout << "\033[1m\033[37m" << totalCores;
            }else std::cout << "\033[1;90m___";
            std::cout << "\033[0m CUDA Cores\n"
                << "     |-> \033[1m\033[37m" << deviceProp.maxThreadsPerMultiProcessor << "\033[0m Threads/MP\n"
                << "         ~> \033[1m\033[37m" << deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor << "\033[0m Total fake Threads" << std::endl;
        }
    }
    std::cout << "\033[1m\033[37m*~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~*\033[0m\n" << std::endl;
    return deviceCount;
}

// --------------------------------- LINEAR ---------------------------------------


void matrixAddition(float* host_A, float* host_B, float* host_C, int rows, int cols) {
    float* dev_A, * dev_B, * dev_C;
    size_t size = rows * cols * sizeof(float);

    HANDLE_ERROR(hipMalloc(&dev_A, size));
    HANDLE_ERROR(hipMalloc(&dev_B, size));
    HANDLE_ERROR(hipMalloc(&dev_C, size));

    HANDLE_ERROR(hipMemcpy(dev_A, host_A, size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_B, host_B, size, hipMemcpyHostToDevice));

    dim3 threads(16, 16);
    dim3 blocks((cols + 15) / 16, (rows + 15) / 16);
    addition <<<blocks, threads >>> (dev_A, dev_B, dev_C, rows, cols);
    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipMemcpy(host_C, dev_C, size, hipMemcpyDeviceToHost));

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
}

void matrixSubtraction(float* host_A, float* host_B, float* host_C, int rows, int cols) {
    float* dev_A, * dev_B, * dev_C;
    size_t size = rows * cols * sizeof(float);

    HANDLE_ERROR(hipMalloc(&dev_A, size));
    HANDLE_ERROR(hipMalloc(&dev_B, size));
    HANDLE_ERROR(hipMalloc(&dev_C, size));

    HANDLE_ERROR(hipMemcpy(dev_A, host_A, size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_B, host_B, size, hipMemcpyHostToDevice));

    dim3 threads(16, 16);
    dim3 blocks((cols + 15) / 16, (rows + 15) / 16);
    addition <<<blocks, threads >>> (dev_A, dev_B, dev_C, rows, cols);
    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipMemcpy(host_C, dev_C, size, hipMemcpyDeviceToHost));

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
}

// --------------------------------- NON LINEAR ---------------------------------------
void matrixMultiplication(float* host_X, float* host_W, float* host_R, int m, int k, int n) {
    float* dev_X, * dev_W, * dev_R;
    size_t sizeX = m * k * sizeof(float);
    size_t sizeW = k * n * sizeof(float);
    size_t sizeR = m * n * sizeof(float);

    HANDLE_ERROR(hipMalloc(&dev_X, sizeX));
    HANDLE_ERROR(hipMalloc(&dev_W, sizeW));
    HANDLE_ERROR(hipMalloc(&dev_R, sizeR));

    HANDLE_ERROR(hipMemcpy(dev_X, host_X, sizeX, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_W, host_W, sizeW, hipMemcpyHostToDevice));

    dim3 threads(16, 16);
    dim3 blocks((n + 15) / 16, (m + 15) / 16);
    multiplication <<<blocks, threads >>> (dev_X, dev_W, dev_R, m, k, n);
    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipMemcpy(host_R, dev_R, sizeR, hipMemcpyDeviceToHost));

    hipFree(dev_X);
    hipFree(dev_W);
    hipFree(dev_R);
}

void matrixDivision(float* host_X, float* host_W, float* host_R, int m, int k, int n) {
    float* dev_X, * dev_W, * dev_R;
    size_t sizeX = m * k * sizeof(float);
    size_t sizeW = k * n * sizeof(float);
    size_t sizeR = m * n * sizeof(float);

    HANDLE_ERROR(hipMalloc(&dev_X, sizeX));
    HANDLE_ERROR(hipMalloc(&dev_W, sizeW));
    HANDLE_ERROR(hipMalloc(&dev_R, sizeR));

    HANDLE_ERROR(hipMemcpy(dev_X, host_X, sizeX, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_W, host_W, sizeW, hipMemcpyHostToDevice));

    dim3 threads(16, 16);
    dim3 blocks((n + 15) / 16, (m + 15) / 16);
    multiplication << <blocks, threads >> > (dev_X, dev_W, dev_R, m, k, n);
    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipMemcpy(host_R, dev_R, sizeR, hipMemcpyDeviceToHost));

    hipFree(dev_X);
    hipFree(dev_W);
    hipFree(dev_R);
}


